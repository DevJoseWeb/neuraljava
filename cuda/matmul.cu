/*
   Copyright 2019 Cleuton Sampaio

   Licensed under the Apache License, Version 2.0 (the "License");
   you may not use this file except in compliance with the License.
   You may obtain a copy of the License at

     http://www.apache.org/licenses/LICENSE-2.0

   Unless required by applicable law or agreed to in writing, software
   distributed under the License is distributed on an "AS IS" BASIS,
   WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
   See the License for the specific language governing permissions and
   limitations under the License.
*/
/* 
    Matrix multiplication sample using CUDA 
    this sample works for squared matrices!
*/

#include "hip/hip_runtime.h"

#include <iostream>
#include <stdio.h>

// CUDA Kernel function: 

__global__ void matmul(float *A, float* B, float *C, int size)
{

    // Row and Column indexes: 
    int row = blockIdx.y*blockDim.y+threadIdx.y;
    int col = blockIdx.x*blockDim.x+threadIdx.x;

    // Are they bellow the maximum?
    if (col < size && row < size) {
       float result = 0;
       for(int ix=0;ix<size;ix++) {
          result += A[row*size+ix]*B[ix*size+col];
       }
       C[row*size+col] = result;
    }

}

int main()
{
    // Matrices and constants
    int size = 3;
    int total = size*size;
    float cpu_A[] = {-1,2,4,0,5,3,6,2,1};
    float cpu_B[] = {3,0,2,3,4,5,4,7,2};
    float cpu_C[total];

    // Allocate device memory:
    float* gpu_A;
    float* gpu_B;
    float* gpu_C;
    int msize = total * sizeof(float);
    hipMalloc((void**)&gpu_A, msize);
    hipMemcpy(gpu_A,cpu_A,msize,hipMemcpyHostToDevice);
    hipMalloc((void**)&gpu_B, msize);
    hipMemcpy(gpu_B,cpu_B,msize,hipMemcpyHostToDevice);
    hipMalloc((void**)&gpu_C,msize);

    // Blocks & grids:
    dim3 blocks(size,size);
    dim3 grid(1,1);

    // Call the kernel:
    matmul<<<grid,blocks>>>(gpu_A,gpu_B,gpu_C,size);

    // Get the result Matrix:
    hipMemcpy(cpu_C,gpu_C,msize,hipMemcpyDeviceToHost);
    std::cout << cpu_C[0] << '\t' << cpu_C[1] << '\t' << cpu_C[2] << std::endl
              << cpu_C[3] << '\t' << cpu_C[4] << '\t' << cpu_C[5] << std::endl
              << cpu_C[6] << '\t' << cpu_C[7] << '\t' << cpu_C[8] << std::endl;

    //Free device matrices
    hipFree(gpu_A);
    hipFree(gpu_B);
    hipFree(gpu_C);
    
}
